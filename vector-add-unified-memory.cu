#include "hip/hip_runtime.h"
#include <stdio.h>

// Luis Miguel García Marín

#define ADD_VECTORS_ITERATIONS 1000  // Define the number of times to execute addVectorsInto

__global__ void initWith(float num, float *a, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride)
    {
        a[i] = num;
    }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride)
    {
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float *vector, int N)
{
    for (int i = 0; i < N; i++)
    {
        if (vector[i] != target)
        {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
            exit(1);
        }
    }
    printf("Success! All values calculated correctly.\n");
}

int main()
{
    int deviceId;
    int numberOfSMs;

    hipSetDevice(0);

    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, 0);

    const int N = 2 << 24;
    size_t size = N * sizeof(float);

    float *a;
    float *b;
    float *c;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // Initialize arrays on CPU first
    for(int i = 0; i < N; i++) {
        a[i] = 3.0f;
        b[i] = 4.0f;
        c[i] = 0.0f;
    }

    // Prefetch to GPU after CPU initialization
    // hipMemPrefetchAsync(a, size, 0);
    // hipMemPrefetchAsync(b, size, 0);
    // hipMemPrefetchAsync(c, size, 0);

    size_t threadsPerBlock;
    size_t numberOfBlocks;

    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;

    hipError_t ignore;
    hipError_t addVectorsErr;
    hipError_t asyncErr;

    ignore = hipGetLastError();

    // No need for initWith kernels since we initialized on CPU
    // initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
    // initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
    // initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);

    // Execute addVectorsInto kernel multiple times
    for (int i = 0; i < ADD_VECTORS_ITERATIONS; i++) {
        addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
        addVectorsErr = hipGetLastError();
        if (addVectorsErr != hipSuccess)
            printf("Error: %s\n", hipGetErrorString(addVectorsErr));

        asyncErr = hipDeviceSynchronize();
        if (asyncErr != hipSuccess)
            printf("Error: %s\n", hipGetErrorString(asyncErr));
    }

    hipMemPrefetchAsync(c, size, hipCpuDeviceId);

    checkElementsAre(7, c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}
